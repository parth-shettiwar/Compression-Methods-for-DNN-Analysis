#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "dnn.hpp"

using namespace std;

//Define the parameters if not defined externally
#ifndef Nn
  #define Nn 1000 // Number of Output Layers
  #define Ni 1000  // Number of Input  Layers
  #define BatchSize 1000
  #define BLOCK_SIZE 1
#endif

float compute_error(float* a, float* b, float size)
{
  float error = 0.0;
  for(int i=0;i<size;i++)
  {
    error += (a[i]-b[i])*(a[i]-b[i]);
  }

  error = sqrt(error/size);
  return error;
}

void fill_classifier(float *synapse, float  *neuron_i, float *neuron_n) {
  for(int n = 0; n < Nn; ++n) {
    for(int i = 0; i < Ni; ++i) {
      synapse[n * Ni + i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX) - 0.5f;
    }
  }

  for(int n = 0; n < Ni; ++n) {
    for(int i = 0; i < BatchSize; ++i) {
      neuron_i[n * BatchSize + i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX) - 0.5f;
    }
  }

  for(int n = 0; n < Nn; ++n) {
    for(int i = 0; i < BatchSize; ++i) {
      neuron_n[n * BatchSize + i] = 0;
    }
  }
}

void classifier_layer( float* synapse, float* neuron_i, float* neuron_n )
{
  for (int n = 0; n < Nn; n++) 
    {
        for (int j = 0; j < BatchSize; ++j) 
        {
            float temp = 0.0;
            for (int i = 0; i < Ni; i++) 
            {
                temp += synapse[n * Ni + i] * neuron_i[i * BatchSize + j];
            }
            neuron_n[n * BatchSize + j] = temp;
        }
    }
}

typedef struct {
    int width;
    int height;
    int stride; 
    float* elements;
} Matrix;

// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col,
                           float value)
{
    A.elements[row * A.stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
 __device__ Matrix GetSubMatrix(float* A, int row, int col, int Awidth) 
{
    Matrix Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = Awidth;
    Asub.elements = &A[Awidth * BLOCK_SIZE * row
                                         + BLOCK_SIZE * col];
    return Asub;
}

__global__ void classifier_layer_batch_cuda(float* synapse, float* neuron_i, float* neuron_n)
{
    // A.width = Ni, B.width = C.width = BatchSize, A = Synapse, B = neuron_i 
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(neuron_n, blockRow, blockCol, BatchSize);

    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    float Cvalue = 0;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
    for (int m = 0; m < (Ni / BLOCK_SIZE); ++m) {

        // Get sub-matrix Asub of A
        Matrix Asub = GetSubMatrix(synapse, blockRow, m, Ni);

        // Get sub-matrix Bsub of B
        Matrix Bsub = GetSubMatrix(neuron_i, m, blockCol, BatchSize);

        // Shared memory used to store Asub and Bsub respectively
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);

        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();
        // Multiply Asub and Bsub together
        for (int e = 0; e < BLOCK_SIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write Csub to device memory
    // Each thread writes one element
    SetElement(Csub, row, col, Cvalue);
}

int main(int argc, char** argv) {

  float * synapse = (float *)malloc(Nn * Ni * sizeof(float));
  float * neuron_i = (float *)malloc(Ni * BatchSize * sizeof(float));
  float * neuron_n = (float *)malloc(Nn * BatchSize * sizeof(float));

  hipError_t err = hipSuccess;

  cout << "initializing arrays\n";

  fill_classifier(synapse,neuron_i,neuron_n);

  cout << "starting computation\n";

  begin_roi();
  classifier_layer(synapse,neuron_i,neuron_n);
  end_roi();

  cout << "simple version complete!\n";  


  // ===========================================================================
  // Allocate the device input vector A
  float *d_synapse = NULL;

  err = hipMalloc((void **)&d_synapse, Nn * Ni * sizeof(float));
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  // Allocate the device input vector B
  float *d_neuron_i = NULL;
  err = hipMalloc((void **)&d_neuron_i, Ni * BatchSize * sizeof(float));

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
  // Allocate the device output vector C
  float *d_neuron_n = NULL;
  err = hipMalloc((void **)&d_neuron_n, Nn * BatchSize * sizeof(float));

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  // for(int i = 0; i < 10; i++)cout << synapse[i] << " ";
  // cout << endl;
  begin_roi();
  // Copy the host input vectors A and B in host memory to the device input vectors in
  // device memory
  printf("Copy input data from the host memory to the CUDA device\n");
  err = hipMemcpy(d_synapse, synapse, Nn * Ni * sizeof(float), hipMemcpyHostToDevice);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipMemcpy(d_neuron_i, neuron_i, Ni * BatchSize * sizeof(float), hipMemcpyHostToDevice);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  int threadsPerBlock = BLOCK_SIZE;
  unsigned int grid_rows = (Nn + threadsPerBlock - 1) / threadsPerBlock;
  unsigned int grid_cols = (BatchSize + threadsPerBlock - 1) / threadsPerBlock;
  dim3 dimGrid(grid_cols, grid_rows);
  dim3 dimBlock(threadsPerBlock, threadsPerBlock);

  
  //int blocksPerGrid =(Nn + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch\n");
  
  classifier_layer_batch_cuda<<<dimGrid, dimBlock>>>(d_synapse, d_neuron_i, d_neuron_n);

  err = hipGetLastError();

  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  float *h_C = (float *)malloc(Nn * BatchSize * sizeof(float));

  // Copy the device result vector in device memory to the host result vector
  // in host memory.
  printf("Copy output data from the CUDA device to the host memory\n");
  err = hipMemcpy(h_C, d_neuron_n, Nn * BatchSize * sizeof(float), hipMemcpyDeviceToHost);


  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }


   // Free device global memory
   err = hipFree(d_synapse);

   if (err != hipSuccess)
   {
       fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
       exit(EXIT_FAILURE);
   }

   err = hipFree(d_neuron_i);

   if (err != hipSuccess)
   {
       fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
       exit(EXIT_FAILURE);
   }

   err = hipFree(d_neuron_n);

   if (err != hipSuccess)
   {
       fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
       exit(EXIT_FAILURE);
   }

   end_roi();

  float error =compute_error(h_C, neuron_n, Nn*BatchSize);
  cout<<"Total error is "<<error<<"\n";

  // Free host memory
  free(synapse);
  free(neuron_i);
  free(neuron_n);

  printf("Done\n");

  return 0;
}

